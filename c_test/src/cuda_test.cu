#include <cmath>
#include <vector>
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <>
#include <cstdio>
#include "cuda_kernel.h"
#include "cuda_err_check.hpp"

struct Vec4 {
    float x, y, z;
    int num;
};

extern  "C" {
void cuda_add_array(const float* const input, float* output, Vec4 vec4) {
    printf("%f, %f, %f, %d\n", vec4.x, vec4.y, vec4.z, vec4.num);
    int block_num = vec4.num >> 8;
    float* input_ptr, *output_ptr;
    CUDA_CHECK_RETURN(hipMalloc((void **) &input_ptr, vec4.num * sizeof(float)));
    CUDA_CHECK_RETURN(hipMalloc((void **) &output_ptr, vec4.num * sizeof(float)));
    CUDA_CHECK_RETURN(hipMemcpy(input_ptr, input, vec4.num * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(output_ptr, output, vec4.num * sizeof(float), hipMemcpyHostToDevice));
    cuda_add_array_kernel<<<block_num, 256>>>(input_ptr, output_ptr, vec4.x, vec4.y, vec4.z);
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    CUDA_CHECK_RETURN(hipMemcpy(output, output_ptr, vec4.num * sizeof(float), hipMemcpyDeviceToHost));
    CUDA_CHECK_RETURN(hipFree(input_ptr));
    CUDA_CHECK_RETURN(hipFree(output_ptr));
}
}
